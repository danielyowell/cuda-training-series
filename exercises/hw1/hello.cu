/*
DANIEL YOWELL
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// this will run on each thread
// we have four threads (2 blocks)
// this will run four times
// the block and thread indices are provided for us
__global__ void hello(){
  // as the lecture says, blockIdx.x and threadIdx.x are provided for us
  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){
  // launch the kernel function
  // 2 blocks, 2 threads
  hello<<<2, 2>>>();
  hipDeviceSynchronize();
}

